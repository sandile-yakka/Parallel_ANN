/*
 * neural_net.c
 *
 *  Created on: Apr 29, 2018
 *      Author: sandile
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>

__const__ int len = 3;


//grid dim == num of input instances
//block dim == num of nodes in layer
__global__ void compute_layer(float* instances, const int len_instance, float* weights, float* out){
	__shared__ float instance[len];
	int bidx = blockIdx.x;
	int tidx = threadIdx.x;
	int tdim = blockDim.x;

	for(unsigned int i = tidx; i < len_instance; i+= tdim){
			instance[i] = instances[ bidx*len_instance + i];
	}
	__syncthreads();
	//All threads have read instance data into memory
	float val = 0.0;
  //dot product
	for(unsigned int i = 0; i < len_instance; i++){
			val += instance[i] * weights[tidx + i*tdim];
	}
	 //apply sigmoid and write output
	out[bidx*tdim + tidx ] = val ; //1.0/(1+exp(-val));
}
// calculate the delta for the outputs
//nb == num of input instances
//npb == num of output nodes

__global__ void delta_j(float* outputs, float* targets, float* deltaJ){

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int tdim = blockDim.x;
	int i = bidx*tdim + tidx;
	printf("%f hello \n", outputs[i]*(1-outputs[i])*(targets[i]-outputs[i]) );
		deltaJ[i] = outputs[i]*(1-outputs[i])*(targets[i]-outputs[i]);

}
//calculates the delta for any hidden layer
//num blocks == num instances
//threads per block == num nodes in layer
__global__ void delta_k(float* layer_outs ,float* deltaJ, int num_outs,
	float* nxt_weights, float* deltaK ){

	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int tdim = blockDim.x;
	int idx = bidx*tdim + tidx;

	float sum = 0.0;
	for(int i = 0; i < num_outs; i++){
		sum += nxt_weights[tidx*num_outs + i]*deltaJ[bidx*num_outs + i];
	}
	deltaK[idx] = layer_outs[idx]*(1 - layer_outs[idx])*sum;
}
//grid dim == number of instances
//threads per block == the total number of weights in the network
__global__ void errDerivates(float* deltaJ, int dj_c, float* deltaK, int dk_c,
		float* in_lay1, int in1_size, float* in_lay2, int in2_size, float* output){

		int tidx = threadIdx.x;
		int bidx = blockIdx.x;
		int tdim = blockDim.x;
		int idx = bidx*tdim + tidx;

		if(tidx < in1_size*in2_size){ // if weight is in hidden layer
			//loop through to find the corresponding delta value
				// for(int i = 1; i <= in2_size; i++){
				// 	if((tidx+1)%i == 0){
				// 		idx_deltak = i-1;
				// 	} paused, gonna explore different logic
				// }
			//
			//so to get corresponding input value if we sayy
			// so if i say the index of the corresponding input is
			int idx_deltak = tidx % in2_size;
			int in_idx = floorf(tidx/in2_size);
			output[idx] = deltaK[bidx*tdim + idx_deltak] * in_lay1[bidx*tdim + in_idx];
		}
		else{
			 //last layer calculations
			 int prev_layer = in1_size * in2_size;
			 int tmp_tidx = tidx - prev_layer;
			 output[idx] = deltaJ[0] * in_lay2[bidx*in2_size + tmp_tidx];
		}
}
//grid dim == num of instances
//threads pb == number of weights
__global__ void reduction_kernel(float* errDerivates, float* output){
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int tdim = blockDim.x;

	atomicAdd(&output[tidx], errDerivates[tidx]);

}
//grid dim == 1 block
//num threads per block == total number of weights in network

__global__ void update_kernel(float* weights, float* new_weights, int lrate, float* deltas){
	int tidx = threadIdx.x;
	int bidx = blockIdx.x;
	int tdim = blockDim.x;
	int idx = bidx*tdim + tidx;

	new_weights[idx] = weights[idx] + lrate*deltas[idx];

}

int main(){

	float* instances = (float*) malloc(6*sizeof(float));
	float* weights = (float*) malloc(9*sizeof(float));
	float* outs = (float*)malloc(6*sizeof(float));

	float* d_instances = 0;
	float* d_weights = 0;
	float* d_out = 0;

	hipMalloc((void**)&d_instances, 6*sizeof(float));
	hipMalloc((void**)&d_weights, 9*sizeof(float));
	hipMalloc((void**)&d_out, 6*sizeof(float));

	for(int i = 0; i < 6; i++){
		instances[i] = 1;
		if(i >= 3){
			instances[i] = 0.5;
		}
	}

	for(int i = 0; i < 9; i++){
		weights[i] = 0.5;
		if(i%3 == 0) weights[i] = 1;
	}

	hipMemcpy(d_instances, instances, 6*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, 9*sizeof(float), hipMemcpyHostToDevice);
	compute_layer<<<2,3>>>(d_instances, 3, d_weights, d_out);

	hipMemcpy(outs, d_out, 6*sizeof(float), hipMemcpyDeviceToHost);
	printf("\n");
	for(int i = 0; i < 6; i++){
		printf("%f ", outs[i]);
	}
	printf("\n");

	// this is for the new layer
	printf("SECOND LAYER ******************\n");

	float* weights2 = (float*) malloc(3*sizeof(float));
	float* outs2 = (float*)malloc(2*sizeof(float));

	for(int i = 0; i < 3; i++) weights2[i] = 1;
	float* dn_instances = 0;
	float* dn_weights = 0;
	hipMalloc((void**)&dn_instances, 6*sizeof(float));
	hipMalloc((void**)&dn_weights, 3*sizeof(float));
	hipMalloc((void**)&d_out, 2*sizeof(float));

	hipMemcpy(dn_instances, outs, 6*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dn_weights, weights2, 3*sizeof(float), hipMemcpyHostToDevice);
	compute_layer<<<2,1>>>(dn_instances, 3, dn_weights, d_out);

	hipMemcpy(outs2, d_out, 2*sizeof(float), hipMemcpyDeviceToHost);
	printf("\n");
	for(int i = 0; i < 2; i++){
		printf("%f ", outs2[i]);
	}
	printf("\n");

	float* targs = 0;
	float* dj = 0;
	hipMalloc((void**)&targs, 2*sizeof(float));
	hipMalloc((void**)&dj, 2*sizeof(float));
	float* targets = (float*) malloc(2*sizeof(float));
	targets[0] = 7;
	targets[1] = 4;



	hipMemcpy(targs, targets, 2*sizeof(float), hipMemcpyHostToDevice);

	delta_j<<<2,1>>>(d_out, targs,dj);
	hipDeviceSynchronize();

	return 0;
}
